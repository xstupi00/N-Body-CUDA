#include "hip/hip_runtime.h"
/**
 * @File nbody.cu
 *
 * Implementation of the N-Body problem
 *
 * Parallel Computations on GPU (PCG 2020)
 * Assignment no. 1 (cuda)
 * Login: xstupi00
 */

#include <cmath>
#include <cfloat>
#include "nbody.h"

/**
 * CUDA kernel to calculate gravitation and collision velocity and update particles.
 * @param p_in    - input particles
 * @param p_out   - output particles
 * @param N       - Number of particles
 * @param dt      - Size of the time step
 */
__global__ void calculate_velocity(t_particles p, int N, float dt)
{
    // Declares dynamic allocation of the shared memory
    extern __shared__ float shared_data[];
    int tx = threadIdx.x;   // thread index
    int bdx = blockDim.x;   // block dimensions
    // Computes the global index of thread within the grid
    int i = bdx * blockIdx.x + tx;

    // Checks whether the thread is not outside the particles borders
    // Loads the particle data covered by the individual thread
    float pos_x = (i < N) ? p.pos_x[i] : 0.0f;
    float pos_y = (i < N) ? p.pos_y[i] : 0.0f;
    float pos_z = (i < N) ? p.pos_z[i] : 0.0f;
    float vel_x = (i < N) ? p.vel_x[i] : 0.0f;
    float vel_y = (i < N) ? p.vel_y[i] : 0.0f;
    float vel_z = (i < N) ? p.vel_z[i] : 0.0f;
    float weight_i = (i < N) ? p.weight[i] : 0.0f;

    // Initialises of auxiliary accumulators of velocity
    float tmp_vel_x = 0.0f;
    float tmp_vel_y = 0.0f;
    float tmp_vel_z = 0.0f;

    // Iterates over the number of blocks within the whole grid
    for (int tile = 0; tile < gridDim.x; tile++) {
        // Computes the index to load the shared particle, which will be used by the whole block
        int idx = tile * bdx + tx;
        int tx7 = tx * 7;   // temporary computation

        // Each thread in the block loads one particle from the global memory to shared memory
        // The whole block will have stored the particles data required in the current iteration
        shared_data[tx7] = (idx < N) ? p.pos_x[idx] : 0.0f;
        shared_data[tx7 + 1] = (idx < N) ? p.pos_y[idx] : 0.0f;
        shared_data[tx7 + 2] = (idx < N) ? p.pos_z[idx] : 0.0f;
        shared_data[tx7 + 3] = (idx < N) ? p.vel_x[idx] : 0.0f;
        shared_data[tx7 + 4] = (idx < N) ? p.vel_y[idx] : 0.0f;
        shared_data[tx7 + 5] = (idx < N) ? p.vel_z[idx] : 0.0f;
        shared_data[tx7 + 6] = (idx < N) ? p.weight[idx] : 0.0f;

        // Waits until each thread within block loads the relevant particle to the shared memory
        __syncthreads();

        // Iterates over the particles which were loaded by whole block
        for (int j = 0; j < bdx; j++) {
            int j7 = j * 7; // temporary calculation

            // Loads the weight of the processing particle
            float weight_j = shared_data[j7 + 6];
            // Instruction Level Parallelism
            float s = -G * dt * weight_j;

            // Computes the distance between the relevant particles
            float r_x = pos_x - shared_data[j7];
            float r_y = pos_y - shared_data[j7 + 1];
            float r_z = pos_z - shared_data[j7 + 2];

            // Computes inverse distance between particles and their distances
            float inv_dist = sqrtf(r_x * r_x + r_y * r_y + r_z * r_z);
            // Computes the gravitation velocity (Fg_dt_m2_r)
            s /= (inv_dist * inv_dist * inv_dist + FLT_MIN);

            // The speed that a particle body receives due to the strength of the relevant particle
            tmp_vel_x += (inv_dist > COLLISION_DISTANCE) ? r_x * s : 0.0f;
            tmp_vel_y += (inv_dist > COLLISION_DISTANCE) ? r_y * s : 0.0f;
            tmp_vel_z += (inv_dist > COLLISION_DISTANCE) ? r_z * s : 0.0f;

            // Checks whether the particles are in the sufficient near distance for collision
            if (inv_dist > 0.0f && inv_dist < COLLISION_DISTANCE) {
                // Computes the temporary partial results to eliminate recalculation
                float weight_diff = weight_i - weight_j;
                float weight_sum = weight_i + weight_j;
                float weight_j_x_2 = 2 * weight_j;

                // Computes the collision velocities between the relevant particles and accumulate the results
                tmp_vel_x += inv_dist > 0.0f ?
                             ((weight_diff * vel_x + weight_j_x_2 * shared_data[j7 + 3]) / weight_sum) - vel_x : 0.0f;
                tmp_vel_y += inv_dist > 0.0f ?
                             ((weight_diff * vel_y + weight_j_x_2 * shared_data[j7 + 4]) / weight_sum) - vel_y : 0.0f;
                tmp_vel_z += inv_dist > 0.0f ?
                             ((weight_diff * vel_z + weight_j_x_2 * shared_data[j7 + 5]) / weight_sum) - vel_z : 0.0f;
            }
        }
        // Waits until each thread within the block terminates the calculation in the current iteration
        __syncthreads();
    }

    // Checks whether the thread is not outside the particles borders
    if (i < N) {
        // Updates the velocity of particles with respect to the computed gravitation and collision velocity
        p.vel_x[i] += tmp_vel_x;
        p.vel_y[i] += tmp_vel_y;
        p.vel_z[i] += tmp_vel_z;

        // Updates the positions of particles with respect to the updated velocity
        p.pos_x[i] += p.vel_x[i] * dt;
        p.pos_y[i] += p.vel_y[i] * dt;
        p.pos_z[i] += p.vel_z[i] * dt;
    }

}// end of calculate_velocity
//---------------------------------------------------------------------------------------------------------------------

/**
 * CUDA kernel to update particles
 * @param p       - particles
 * @param comX    - pointer to a center of mass position in X
 * @param comY    - pointer to a center of mass position in Y
 * @param comZ    - pointer to a center of mass position in Z
 * @param comW    - pointer to a center of mass weight
 * @param lock    - pointer to a user-implemented lock
 * @param N       - Number of particles
 */
__global__ void centerOfMass(t_particles p, float* comX, float* comY, float* comZ, float* comW, int* lock, const int N)
{

}// end of centerOfMass
//----------------------------------------------------------------------------------------------------------------------

/**
 * CPU implementation of the Center of Mass calculation
 * @param particles - All particles in the system
 * @param N         - Number of particles
 */
__host__ float4 centerOfMassCPU(MemDesc& memDesc)
{
  float4 com = {0 ,0, 0, 0};

  for(int i = 0; i < memDesc.getDataSize(); i++)
  {
    // Calculate the vector on the line connecting points and most recent position of center-of-mass
    const float dx = memDesc.getPosX(i) - com.x;
    const float dy = memDesc.getPosY(i) - com.y;
    const float dz = memDesc.getPosZ(i) - com.z;

    // Calculate weight ratio only if at least one particle isn't massless
    const float dw = ((memDesc.getWeight(i) + com.w) > 0.0f)
                          ? ( memDesc.getWeight(i) / (memDesc.getWeight(i) + com.w)) : 0.0f;

    // Update position and weight of the center-of-mass according to the weight ration and vector
    com.x += dx * dw;
    com.y += dy * dw;
    com.z += dz * dw;
    com.w += memDesc.getWeight(i);
  }
  return com;
}// enf of centerOfMassCPU
//----------------------------------------------------------------------------------------------------------------------
