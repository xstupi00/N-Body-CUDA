#include "hip/hip_runtime.h"
/**
 * @File main.cu
 *
 * The main file of the project
 *
 * Parallel Computations on GPU (PCG 2020)
 * Assignment no. 1 (cuda)
 * Login: xstupi00
 */

#include <sys/time.h>
#include <cstdio>
#include <cmath>

#include "nbody.h"
#include "h5Helper.h"


#define checkCudaErrors(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}


/**
 * Main routine
 * @param argc
 * @param argv
 * @return
 */
int main(int argc, char **argv) {
    // Time measurement
    struct timeval t1, t2;

    if (argc != 10) {
        printf("Usage: "
               "nbody <N> <dt> <steps> <threads/block> <write intesity> "
               "<reduction threads> <reduction threads/block> <input> <output>\n"
        );
        exit(1);
    }

    // Number of particles
    const int N = std::stoi(argv[1]);
    // Length of time step
    const float dt = std::stof(argv[2]);
    // Number of steps
    const int steps = std::stoi(argv[3]);
    // Number of thread blocks
    const int thr_blc = std::stoi(argv[4]);
    // Write frequency
    int writeFreq = std::stoi(argv[5]);
    // number of reduction threads
    const int red_thr = std::stoi(argv[6]);
    // Number of reduction threads/blocks
    const int red_thr_blc = std::stoi(argv[7]);

    // Size of the simulation CUDA gird - number of blocks
    const size_t simulationGrid = (N + thr_blc - 1) / thr_blc;
    // Size of the reduction CUDA grid - number of blocks
    const size_t reductionGrid = (red_thr + red_thr_blc - 1) / red_thr_blc;

    // Log benchmark setup
    printf("N: %d\n", N);
    printf("dt: %f\n", dt);
    printf("steps: %d\n", steps);
    printf("threads/block: %d\n", thr_blc);
    printf("blocks/grid: %lu\n", simulationGrid);
    printf("reduction threads/block: %d\n", red_thr_blc);
    printf("reduction blocks/grid: %lu\n", reductionGrid);

    // Number of records to continuous writing of partial results
    const size_t recordsNum = (writeFreq > 0) ? (steps + writeFreq - 1) / writeFreq : 0;
    writeFreq = (writeFreq > 0) ? writeFreq : 0;

    // CPU particles structures
    t_particles particles_cpu;

    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    //                            FILL IN: CPU side memory allocation (step 0)                                        //
    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    // The overall memory size of input particles
    size_t size = N * sizeof(float);
    // Allocates page-locked memory on the host. Maps the allocation into the CUDA address space
    checkCudaErrors(hipHostAlloc(&particles_cpu.pos_x, size, hipHostMallocMapped));
    checkCudaErrors(hipHostAlloc(&particles_cpu.pos_y, size, hipHostMallocMapped));
    checkCudaErrors(hipHostAlloc(&particles_cpu.pos_z, size, hipHostMallocMapped));
    checkCudaErrors(hipHostAlloc(&particles_cpu.vel_x, size, hipHostMallocMapped));
    checkCudaErrors(hipHostAlloc(&particles_cpu.vel_y, size, hipHostMallocMapped));
    checkCudaErrors(hipHostAlloc(&particles_cpu.vel_z, size, hipHostMallocMapped));
    checkCudaErrors(hipHostAlloc(&particles_cpu.weight, size, hipHostMallocMapped));

    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    //                              FILL IN: memory layout descriptor (step 0)                                        //
    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    /*
     * Caution! Create only after CPU side allocation
     * parameters:
     *                      Stride of two               Offset of the first
     *  Data pointer        consecutive elements        element in floats,
     *                      in floats, not bytes        not bytes
    */
    MemDesc md(
            particles_cpu.pos_x, 1, 0,              // Position in X
            particles_cpu.pos_y, 1, 0,              // Position in Y
            particles_cpu.pos_z, 1, 0,              // Position in Z
            particles_cpu.vel_x, 1, 0,              // Velocity in X
            particles_cpu.vel_y, 1, 0,              // Velocity in Y
            particles_cpu.vel_z, 1, 0,              // Velocity in Z
            particles_cpu.weight, 1, 0,             // Weight
            N,                                      // Number of particles
            recordsNum                              // Number of records in output file
    );

    // Initialisation of helper class and loading of input data
    H5Helper h5Helper(argv[8], argv[9], md);

    try {
        h5Helper.init();
        h5Helper.readParticleData();
    }
    catch (const std::exception &e) {
        std::cerr << e.what() << std::endl;
        return -1;
    }

    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    //                                  FILL IN: GPU side memory allocation (step 0)                                  //
    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    // GPU particles structure
    t_particles particles_gpu;
    // GPU auxiliary velocities structure
    t_velocities tmp_vel;

    // Allocate memory on the device
    checkCudaErrors(hipMalloc(&particles_gpu.pos_x, size));
    checkCudaErrors(hipMalloc(&particles_gpu.pos_y, size));
    checkCudaErrors(hipMalloc(&particles_gpu.pos_z, size));
    checkCudaErrors(hipMalloc(&particles_gpu.vel_x, size));
    checkCudaErrors(hipMalloc(&particles_gpu.vel_y, size));
    checkCudaErrors(hipMalloc(&particles_gpu.vel_z, size));
    checkCudaErrors(hipMalloc(&particles_gpu.weight, size));
    checkCudaErrors(hipMalloc(&tmp_vel.x, size));
    checkCudaErrors(hipMalloc(&tmp_vel.y, size));
    checkCudaErrors(hipMalloc(&tmp_vel.z, size));

    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    //                                       FILL IN: memory transfers (step 0)                                       //
    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

    // Copies particles data from host to device.
    checkCudaErrors(hipMemcpy(particles_gpu.pos_x, particles_cpu.pos_x, size, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(particles_gpu.pos_y, particles_cpu.pos_y, size, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(particles_gpu.pos_z, particles_cpu.pos_z, size, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(particles_gpu.vel_x, particles_cpu.vel_x, size, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(particles_gpu.vel_y, particles_cpu.vel_y, size, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(particles_gpu.vel_z, particles_cpu.vel_z, size, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(particles_gpu.weight, particles_cpu.weight, size, hipMemcpyHostToDevice));

    gettimeofday(&t1, 0);

    for (int s = 0; s < steps; s++) {
        ////////////////////////////////////////////////////////////////////////////////////////////////////////////////
        //                                       FILL IN: kernels invocation (step 0)                                 //
        ////////////////////////////////////////////////////////////////////////////////////////////////////////////////

        calculate_gravitation_velocity << < simulationGrid, thr_blc >> > (particles_gpu, tmp_vel, N, dt);
        calculate_collision_velocity << < simulationGrid, thr_blc >> > (particles_gpu, tmp_vel, N, dt);
        update_particle << < simulationGrid, thr_blc >> > (particles_gpu, tmp_vel, N, dt);

        ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
        //                                          FILL IN: synchronization  (step 4)                                    //
        ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

        if (writeFreq > 0 && (s % writeFreq == 0)) {
            //////////////////////////////////////////////////////////////////////////////////////////////////////////////////
            //                          FILL IN: synchronization and file access logic (step 4)                             //
            //////////////////////////////////////////////////////////////////////////////////////////////////////////////////
        }
    }

    //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    //              FILL IN: invocation of center-of-mass kernel (step 3.1, step 3.2, step 4)                           //
    //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    hipDeviceSynchronize();

    gettimeofday(&t2, 0);

    // Approximate simulation wall time
    double t = (1000000.0 * (t2.tv_sec - t1.tv_sec) + t2.tv_usec - t1.tv_usec) / 1000000.0;
    printf("Time: %f s\n", t);

    //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    //                             FILL IN: memory transfers for particle data (step 0)                                 //
    //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    float4 comOnGPU;

    // Copies particles data from device to host.
    checkCudaErrors(hipMemcpy(particles_cpu.pos_x, particles_gpu.pos_x, size, hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(particles_cpu.pos_y, particles_gpu.pos_y, size, hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(particles_cpu.pos_z, particles_gpu.pos_z, size, hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(particles_cpu.vel_x, particles_gpu.vel_x, size, hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(particles_cpu.vel_y, particles_gpu.vel_y, size, hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(particles_cpu.vel_z, particles_gpu.vel_z, size, hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(particles_cpu.weight, particles_gpu.weight, size, hipMemcpyDeviceToHost));

    //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    //                        FILL IN: memory transfers for center-of-mass (step 3.1, step 3.2)                         //
    //////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    float4 comOnCPU = centerOfMassCPU(md);

    std::cout << "Center of mass on CPU:" << std::endl
              << comOnCPU.x << ", "
              << comOnCPU.y << ", "
              << comOnCPU.z << ", "
              << comOnCPU.w
              << std::endl;

    std::cout << "Center of mass on GPU:" << std::endl
              << comOnGPU.x << ", "
              << comOnGPU.y << ", "
              << comOnGPU.z << ", "
              << comOnGPU.w
              << std::endl;

    // Writing final values to the file
    h5Helper.writeComFinal(comOnGPU.x, comOnGPU.y, comOnGPU.z, comOnGPU.w);
    h5Helper.writeParticleDataFinal();

    // Free page-locked memory.
    checkCudaErrors(hipHostFree(particles_cpu.pos_x));
    checkCudaErrors(hipHostFree(particles_cpu.pos_y));
    checkCudaErrors(hipHostFree(particles_cpu.pos_z));
    checkCudaErrors(hipHostFree(particles_cpu.vel_x));
    checkCudaErrors(hipHostFree(particles_cpu.vel_y));
    checkCudaErrors(hipHostFree(particles_cpu.vel_z));
    checkCudaErrors(hipHostFree(particles_cpu.weight));

    // Free memory on the device.
    checkCudaErrors(hipFree(particles_gpu.pos_x));
    checkCudaErrors(hipFree(particles_gpu.pos_y));
    checkCudaErrors(hipFree(particles_gpu.pos_z));
    checkCudaErrors(hipFree(particles_gpu.vel_x));
    checkCudaErrors(hipFree(particles_gpu.vel_y));
    checkCudaErrors(hipFree(particles_gpu.vel_z));
    checkCudaErrors(hipFree(particles_gpu.weight));

    return 0;
}// end of main
//----------------------------------------------------------------------------------------------------------------------
