#include "hip/hip_runtime.h"
/**
 * @File nbody.cu
 *
 * Implementation of the N-Body problem
 *
 * Parallel Computations on GPU (PCG 2020)
 * Assignment no. 1 (cuda)
 * Login: xstupi00
 */

#include <cmath>
#include <cfloat>
#include "nbody.h"

__global__ void calculate_velocity(t_particles p, int N, float dt)
{
    extern __shared__ float shared_data[];
    int tx = threadIdx.x;
    int bdx = blockDim.x;
    int i = bdx * blockIdx.x + tx;

    float pos_x = (i < N) ? p.pos_x[i] : 0.0f;
    float pos_y = (i < N) ? p.pos_y[i] : 0.0f;
    float pos_z = (i < N) ? p.pos_z[i] : 0.0f;
    float vel_x = (i < N) ? p.vel_x[i] : 0.0f;
    float vel_y = (i < N) ? p.vel_y[i] : 0.0f;
    float vel_z = (i < N) ? p.vel_y[i] : 0.0f;
    float weight_i = (i < N) ? p.weight[i] : 0.0f;

    float tmp_vel_x = 0.0f;
    float tmp_vel_y = 0.0f;
    float tmp_vel_z = 0.0f;

    for (int tile = 0; tile < gridDim.x; tile++) {
        int idx = tile * bdx + tx;
        int tx7 = tx * 7;

        shared_data[tx7] = (idx < N) ? p.pos_x[idx] : 0.0f;
        shared_data[tx7 + 1] = (idx < N) ? p.pos_y[idx] : 0.0f;
        shared_data[tx7 + 2] = (idx < N) ? p.pos_z[idx] : 0.0f;
        shared_data[tx7 + 3] = (idx < N) ? p.vel_x[idx] : 0.0f;
        shared_data[tx7 + 4] = (idx < N) ? p.vel_y[idx] : 0.0f;
        shared_data[tx7 + 5] = (idx < N) ? p.vel_z[idx] : 0.0f;
        shared_data[tx7 + 6] = (idx < N) ? p.weight[idx] : 0.0f;

        __syncthreads();

        for (int j = 0; j < bdx; j++) {
            int j7 = j * 7;

            float r_x = pos_x - shared_data[j7];
            float r_y = pos_y - shared_data[j7 + 1];
            float r_z = pos_z - shared_data[j7 + 2];

            float weight_j = shared_data[j7 + 6];
            float inv_dist = sqrtf(r_x * r_x + r_y * r_y + r_z * r_z);
            float s = weight_j * (-G * dt / (inv_dist * inv_dist * inv_dist + FLT_MIN));

            tmp_vel_x += (inv_dist > COLLISION_DISTANCE) ? r_x * s : 0.0f;
            tmp_vel_y += (inv_dist > COLLISION_DISTANCE) ? r_y * s : 0.0f;
            tmp_vel_z += (inv_dist > COLLISION_DISTANCE) ? r_z * s : 0.0f;

            if (inv_dist > 0.0f && inv_dist < COLLISION_DISTANCE) {

                float weight_diff = weight_i - weight_j;
                float weight_sum = weight_i + weight_j;
                float weight_j_x_2 = 2 * weight_j;

                tmp_vel_x += inv_dist > 0.0f ?
                             ((weight_diff * vel_x + weight_j_x_2 * shared_data[j7 + 3]) / weight_sum) - vel_x : 0.0f;
                tmp_vel_y += inv_dist > 0.0f ?
                             ((weight_diff * vel_y + weight_j_x_2 * shared_data[j7 + 4]) / weight_sum) - vel_y : 0.0f;
                tmp_vel_z += inv_dist > 0.0f ?
                             ((weight_diff * vel_z + weight_j_x_2 * shared_data[j7 + 5]) / weight_sum) - vel_z : 0.0f;
            }
        }
        __syncthreads();
    }

    if (i < N) {
        p.vel_x[i] += tmp_vel_x;
        p.vel_y[i] += tmp_vel_y;
        p.vel_z[i] += tmp_vel_z;

        p.pos_x[i] += p.vel_x[i] * dt;
        p.pos_y[i] += p.vel_y[i] * dt;
        p.pos_z[i] += p.vel_z[i] * dt;
    }

}// end of calculate_velocity
//---------------------------------------------------------------------------------------------------------------------

/**
 * CPU implementation of the Center of Mass calculation
 * @param particles - All particles in the system
 * @param N         - Number of particles
 */
__host__ float4 centerOfMassCPU(MemDesc& memDesc)
{
  float4 com = {0 ,0, 0, 0};

  for(int i = 0; i < memDesc.getDataSize(); i++)
  {
    // Calculate the vector on the line connecting points and most recent position of center-of-mass
    const float dx = memDesc.getPosX(i) - com.x;
    const float dy = memDesc.getPosY(i) - com.y;
    const float dz = memDesc.getPosZ(i) - com.z;

    // Calculate weight ratio only if at least one particle isn't massless
    const float dw = ((memDesc.getWeight(i) + com.w) > 0.0f)
                          ? ( memDesc.getWeight(i) / (memDesc.getWeight(i) + com.w)) : 0.0f;

    // Update position and weight of the center-of-mass according to the weight ration and vector
    com.x += dx * dw;
    com.y += dy * dw;
    com.z += dz * dw;
    com.w += memDesc.getWeight(i);
  }
  return com;
}// enf of centerOfMassCPU
//----------------------------------------------------------------------------------------------------------------------
