#include "hip/hip_runtime.h"
/**
 * @File nbody.cu
 *
 * Implementation of the N-Body problem
 *
 * Parallel Computations on GPU (PCG 2020)
 * Assignment no. 1 (cuda)
 * Login: xstupi00
 */

#include <cmath>
#include <cfloat>
#include "nbody.h"

/**
 * CUDA kernel to calculate gravitation and collision velocity and update particles.
 * @param p_in    - input particles
 * @param p_out   - output particles
 * @param N       - Number of particles
 * @param dt      - Size of the time step
 */
__global__ void calculate_velocity(
        const t_particles p_in, t_particles p_out, int N, float dt
) {
    // Computes the global index of thread within the grid
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    // Checks whether the thread is not outside the particles borders
    if (i < N) {

        // Loads the particle data covered by the individual thread
        float pos_x = p_in.pos_x[i];
        float pos_y = p_in.pos_y[i];
        float pos_z = p_in.pos_z[i];
        float vel_x = p_in.vel_x[i];
        float vel_y = p_in.vel_y[i];
        float vel_z = p_in.vel_z[i];
        float weight_i = p_in.weight[i];

        // Initialises of auxiliary accumulators of velocity
        float tmp_vel_x = 0.0f;
        float tmp_vel_y = 0.0f;
        float tmp_vel_z = 0.0f;

        // The iterations over all particles to compute the gravitation velocity to them
        for (int j = 0; j < N; j++) {

            // Loads the weight of the processing particle
            float weight_j = p_in.weight[j];
            // Instruction Level Parallelism
            float s = -G * dt * weight_j;

            // Computes the distance between the relevant particles
            float r_x = pos_x - p_in.pos_x[j];
            float r_y = pos_y - p_in.pos_y[j];
            float r_z = pos_z - p_in.pos_z[j];

            // Computes inverse distance between particles and their distances
            float inv_dist = sqrtf(r_x * r_x + r_y * r_y + r_z * r_z);
            // Computes the gravitation velocity (Fg_dt_m2_r)
            s /= (inv_dist * inv_dist * inv_dist + FLT_MIN);

            // The speed that a particle body receives due to the strength of the relevant particle
            tmp_vel_x += (inv_dist > COLLISION_DISTANCE) ? r_x * s : 0.0f;
            tmp_vel_y += (inv_dist > COLLISION_DISTANCE) ? r_y * s : 0.0f;
            tmp_vel_z += (inv_dist > COLLISION_DISTANCE) ? r_z * s : 0.0f;

            // Checks whether the particles are in the sufficient near distance for collision
            if (inv_dist > 0.0f && inv_dist < COLLISION_DISTANCE) {
                // Computes the temporary partial results to eliminate recalculation
                float weight_diff = weight_i - weight_j;
                float weight_sum = weight_i + weight_j;
                float weight_j_x_2 = 2 * weight_j;

                // Computes the collision velocities between the relevant particles and accumulate the results
                tmp_vel_x += inv_dist > 0.0f ?
                             ((weight_diff * vel_x + weight_j_x_2 * p_in.vel_x[j]) / weight_sum) - vel_x : 0.0f;
                tmp_vel_y += inv_dist > 0.0f ?
                             ((weight_diff * vel_y + weight_j_x_2 * p_in.vel_y[j]) / weight_sum) - vel_y : 0.0f;
                tmp_vel_z += inv_dist > 0.0f ?
                             ((weight_diff * vel_z + weight_j_x_2 * p_in.vel_z[j]) / weight_sum) - vel_z : 0.0f;
            }
        }

        // Pre-computes new velocity
        vel_x += tmp_vel_x;
        vel_y += tmp_vel_y;
        vel_z += tmp_vel_z;

        // Updates the velocity of particles with respect to the computed gravitation and collision velocity
        p_out.vel_x[i] = vel_x;
        p_out.vel_y[i] = vel_y;
        p_out.vel_z[i] = vel_z;

        // Updates the positions of particles with respect to the updated velocity
        p_out.pos_x[i] = pos_x + vel_x * dt;
        p_out.pos_y[i] = pos_y + vel_y * dt;
        p_out.pos_z[i] = pos_z + vel_z * dt;
    }

}// end of calculate_velocity
//---------------------------------------------------------------------------------------------------------------------

/**
 * CUDA kernel to update particles
 * @param p       - particles
 * @param comX    - pointer to a center of mass position in X
 * @param comY    - pointer to a center of mass position in Y
 * @param comZ    - pointer to a center of mass position in Z
 * @param comW    - pointer to a center of mass weight
 * @param lock    - pointer to a user-implemented lock
 * @param N       - Number of particles
 */
__global__ void centerOfMass(t_particles p, float* comX, float* comY, float* comZ, float* comW, int* lock, const int N)
{

}// end of centerOfMass
//----------------------------------------------------------------------------------------------------------------------

/**
 * CPU implementation of the Center of Mass calculation
 * @param particles - All particles in the system
 * @param N         - Number of particles
 */
__host__ float4 centerOfMassCPU(MemDesc& memDesc)
{
  float4 com = {0 ,0, 0, 0};

  for(int i = 0; i < memDesc.getDataSize(); i++)
  {
    // Calculate the vector on the line connecting points and most recent position of center-of-mass
    const float dx = memDesc.getPosX(i) - com.x;
    const float dy = memDesc.getPosY(i) - com.y;
    const float dz = memDesc.getPosZ(i) - com.z;

    // Calculate weight ratio only if at least one particle isn't massless
    const float dw = ((memDesc.getWeight(i) + com.w) > 0.0f)
                          ? ( memDesc.getWeight(i) / (memDesc.getWeight(i) + com.w)) : 0.0f;

    // Update position and weight of the center-of-mass according to the weight ration and vector
    com.x += dx * dw;
    com.y += dy * dw;
    com.z += dz * dw;
    com.w += memDesc.getWeight(i);
  }
  return com;
}// enf of centerOfMassCPU
//----------------------------------------------------------------------------------------------------------------------
