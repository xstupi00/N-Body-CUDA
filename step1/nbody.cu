#include "hip/hip_runtime.h"
/**
 * @File nbody.cu
 *
 * Implementation of the N-Body problem
 *
 * Parallel Computations on GPU (PCG 2020)
 * Assignment no. 1 (cuda)
 * Login: xstupi00
 */

#include <cmath>
#include <cfloat>
#include "nbody.h"

/**
 * CUDA kernel to calculate gravitation and collision velocity and update particles.
 * @param p_in    - input particles
 * @param p_out   - output particles
 * @param N       - Number of particles
 * @param dt      - Size of the time step
 */
__global__ void calculate_velocity(
        const t_particles p_in, t_particles p_out, int N, float dt
) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < N) {

        float pos_x = p_in.pos_x[i];
        float pos_y = p_in.pos_y[i];
        float pos_z = p_in.pos_z[i];
        float vel_x = p_in.vel_x[i];
        float vel_y = p_in.vel_y[i];
        float vel_z = p_in.vel_z[i];
        float weight_i = p_in.weight[i];

        float tmp_vel_x = 0.0f;
        float tmp_vel_y = 0.0f;
        float tmp_vel_z = 0.0f;

        for (int j = 0; j < N; j++) {

            float r_x = pos_x - p_in.pos_x[j];
            float r_y = pos_y - p_in.pos_y[j];
            float r_z = pos_z - p_in.pos_z[j];

            float weight_j = p_in.weight[j];
            float inv_dist = sqrtf(r_x * r_x + r_y * r_y + r_z * r_z);
            float s = weight_j * (-G * dt / (inv_dist * inv_dist * inv_dist + FLT_MIN));

            tmp_vel_x += (inv_dist > COLLISION_DISTANCE) ? r_x * s : 0.0f;
            tmp_vel_y += (inv_dist > COLLISION_DISTANCE) ? r_y * s : 0.0f;
            tmp_vel_z += (inv_dist > COLLISION_DISTANCE) ? r_z * s : 0.0f;

            if (inv_dist > 0.0f && inv_dist < COLLISION_DISTANCE) {

                float weight_diff = weight_i - weight_j;
                float weight_sum = weight_i + weight_j;
                float weight_j_x_2 = 2 * weight_j;

                tmp_vel_x += inv_dist > 0.0f ?
                             ((weight_diff * vel_x + weight_j_x_2 * p_in.vel_x[j]) / weight_sum) - vel_x : 0.0f;
                tmp_vel_y += inv_dist > 0.0f ?
                             ((weight_diff * vel_y + weight_j_x_2 * p_in.vel_y[j]) / weight_sum) - vel_y : 0.0f;
                tmp_vel_z += inv_dist > 0.0f ?
                             ((weight_diff * vel_z + weight_j_x_2 * p_in.vel_z[j]) / weight_sum) - vel_z : 0.0f;
            }
        }

        p_out.vel_x[i] = vel_x + tmp_vel_x;
        p_out.vel_y[i] = vel_y + tmp_vel_y;
        p_out.vel_z[i] = vel_z + tmp_vel_z;

        p_out.pos_x[i] = pos_x + p_out.vel_x[i] * dt;
        p_out.pos_y[i] = pos_y + p_out.vel_y[i] * dt;
        p_out.pos_z[i] = pos_z + p_out.vel_z[i] * dt;
    }

}// end of calculate_velocity
//---------------------------------------------------------------------------------------------------------------------

/**
 * CUDA kernel to update particles
 * @param p       - particles
 * @param comX    - pointer to a center of mass position in X
 * @param comY    - pointer to a center of mass position in Y
 * @param comZ    - pointer to a center of mass position in Z
 * @param comW    - pointer to a center of mass weight
 * @param lock    - pointer to a user-implemented lock
 * @param N       - Number of particles
 */
__global__ void centerOfMass(t_particles p, float* comX, float* comY, float* comZ, float* comW, int* lock, const int N)
{

}// end of centerOfMass
//----------------------------------------------------------------------------------------------------------------------

/**
 * CPU implementation of the Center of Mass calculation
 * @param particles - All particles in the system
 * @param N         - Number of particles
 */
__host__ float4 centerOfMassCPU(MemDesc& memDesc)
{
  float4 com = {0 ,0, 0, 0};

  for(int i = 0; i < memDesc.getDataSize(); i++)
  {
    // Calculate the vector on the line connecting points and most recent position of center-of-mass
    const float dx = memDesc.getPosX(i) - com.x;
    const float dy = memDesc.getPosY(i) - com.y;
    const float dz = memDesc.getPosZ(i) - com.z;

    // Calculate weight ratio only if at least one particle isn't massless
    const float dw = ((memDesc.getWeight(i) + com.w) > 0.0f)
                          ? ( memDesc.getWeight(i) / (memDesc.getWeight(i) + com.w)) : 0.0f;

    // Update position and weight of the center-of-mass according to the weight ration and vector
    com.x += dx * dw;
    com.y += dy * dw;
    com.z += dz * dw;
    com.w += memDesc.getWeight(i);
  }
  return com;
}// enf of centerOfMassCPU
//----------------------------------------------------------------------------------------------------------------------
