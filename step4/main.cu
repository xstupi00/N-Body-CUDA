#include "hip/hip_runtime.h"
/**
 * @File main.cu
 *
 * The main file of the project
 *
 * Parallel Computations on GPU (PCG 2020)
 * Assignment no. 1 (cuda)
 * Login: xstupi00
 */

#include <sys/time.h>
#include <cstdio>
#include <cmath>
#include <vector>

#include <hip/hip_runtime_api.h>

#include "nbody.h"
#include "h5Helper.h"


/**
 * Main rotine
 * @param argc
 * @param argv
 * @return
 */
int main(int argc, char **argv) {
    // Time measurement
    struct timeval t1, t2;

    if (argc != 10) {
        printf("Usage: nbody <N> <dt> <steps> <threads/block> <write intesity> <reduction threads> <reduction threads/block> <input> <output>\n");
        exit(1);
    }

    // Number of particles
    const int N = std::stoi(argv[1]);
    // Length of time step
    const float dt = std::stof(argv[2]);
    // Number of steps
    const int steps = std::stoi(argv[3]);
    // Number of thread blocks
    const int thr_blc = std::stoi(argv[4]);
    // Write frequency
    int writeFreq = std::stoi(argv[5]);
    // number of reduction threads
    const int red_thr = std::stoi(argv[6]);
    // Number of reduction threads/blocks
    const int red_thr_blc = std::stoi(argv[7]);

    // Size of the simulation CUDA gird - number of blocks
    const size_t simulationGrid = (N + thr_blc - 1) / thr_blc;
    // Size of the reduction CUDA grid - number of blocks
    const size_t reductionGrid = (red_thr + red_thr_blc - 1) / red_thr_blc;

    // Log benchmark setup
    printf("N: %d\n", N);
    printf("dt: %f\n", dt);
    printf("steps: %d\n", steps);
    printf("threads/block: %d\n", thr_blc);
    printf("blocks/grid: %lu\n", simulationGrid);
    printf("reduction threads/block: %d\n", red_thr_blc);
    printf("reduction blocks/grid: %lu\n", reductionGrid);

    const size_t recordsNum = (writeFreq > 0) ? (steps + writeFreq - 1) / writeFreq : 0;
    writeFreq = (writeFreq > 0) ? writeFreq : 0;


    t_particles particles_cpu;

    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    //                            FILL IN: CPU side memory allocation (step 0)                                        //
    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    size_t size = N * sizeof(float);
    checkCudaErrors(hipHostAlloc(&particles_cpu.pos_x, size, hipHostMallocMapped));
    checkCudaErrors(hipHostAlloc(&particles_cpu.pos_y, size, hipHostMallocMapped));
    checkCudaErrors(hipHostAlloc(&particles_cpu.pos_z, size, hipHostMallocMapped));
    checkCudaErrors(hipHostAlloc(&particles_cpu.vel_x, size, hipHostMallocMapped));
    checkCudaErrors(hipHostAlloc(&particles_cpu.vel_y, size, hipHostMallocMapped));
    checkCudaErrors(hipHostAlloc(&particles_cpu.vel_z, size, hipHostMallocMapped));
    checkCudaErrors(hipHostAlloc(&particles_cpu.weight, size, hipHostMallocMapped));

    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    //                              FILL IN: memory layout descriptor (step 0)                                        //
    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    /*
     * Caution! Create only after CPU side allocation
     * parameters:
     *                      Stride of two               Offset of the first
     *  Data pointer        consecutive elements        element in floats,
     *                      in floats, not bytes        not bytes
    */
    MemDesc md(
            particles_cpu.pos_x, 1, 0,              // Position in X
            particles_cpu.pos_y, 1, 0,              // Position in Y
            particles_cpu.pos_z, 1, 0,              // Position in Z
            particles_cpu.vel_x, 1, 0,              // Velocity in X
            particles_cpu.vel_y, 1, 0,              // Velocity in Y
            particles_cpu.vel_z, 1, 0,              // Velocity in Z
            particles_cpu.weight, 1, 0,             // Weight
            N,                                      // Number of particles
            recordsNum                              // Number of records in output file
    );

    // Initialisation of helper class and loading of input data
    H5Helper h5Helper(argv[8], argv[9], md);

    try {
        h5Helper.init();
        h5Helper.readParticleData();
    }
    catch (const std::exception &e) {
        std::cerr << e.what() << std::endl;
        return -1;
    }

    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    //                                  FILL IN: GPU side memory allocation (step 0)                                  //
    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    std::vector<t_particles> particles_gpu(2);

    for (auto &p_gpu : particles_gpu) {
        checkCudaErrors(hipMalloc(&p_gpu.pos_x, size));
        checkCudaErrors(hipMalloc(&p_gpu.pos_y, size));
        checkCudaErrors(hipMalloc(&p_gpu.pos_z, size));
        checkCudaErrors(hipMalloc(&p_gpu.vel_x, size));
        checkCudaErrors(hipMalloc(&p_gpu.vel_y, size));
        checkCudaErrors(hipMalloc(&p_gpu.vel_z, size));
        checkCudaErrors(hipMalloc(&p_gpu.weight, size));
    }

    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    //                                       FILL IN: memory transfers (step 0)                                       //
    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

    checkCudaErrors(hipMemcpy(particles_gpu[0].pos_x, particles_cpu.pos_x, size, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(particles_gpu[0].pos_y, particles_cpu.pos_y, size, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(particles_gpu[0].pos_z, particles_cpu.pos_z, size, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(particles_gpu[0].vel_x, particles_cpu.vel_x, size, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(particles_gpu[0].vel_y, particles_cpu.vel_y, size, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(particles_gpu[0].vel_z, particles_cpu.vel_z, size, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(particles_gpu[0].weight, particles_cpu.weight, size, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(particles_gpu[1].weight, particles_gpu[0].weight, size, hipMemcpyDeviceToDevice));

    float4* comCPU;
    float4* comGPU;
    int* lock;

    checkCudaErrors(hipHostAlloc(&comCPU, sizeof(float4), hipHostMallocMapped));
    checkCudaErrors(hipMalloc(&comGPU, sizeof(float4)));
    checkCudaErrors(hipMalloc(&lock, sizeof(int)));

    checkCudaErrors(hipMemset(comCPU, 0, sizeof(float4)));
    checkCudaErrors(hipMemset(lock, 0, sizeof(int)));

    size_t shm_mem_calc = thr_blc * sizeof(float) * 7;
    size_t shm_mem_mass = (red_thr_blc <= 32) ? 2 * red_thr_blc * sizeof(float) * 4 : red_thr_blc * sizeof(float) * 4;

    hipStream_t cm_stream, cp_stream, wp_stream;
    checkCudaErrors(hipStreamCreate(&cm_stream));
    checkCudaErrors(hipStreamCreate(&cp_stream));
    checkCudaErrors(hipStreamCreate(&wp_stream));

    hipEvent_t cm_event, cp_event, wp_event;
    checkCudaErrors(hipEventCreate(&cm_event));
    checkCudaErrors(hipEventCreate(&cp_event));
    checkCudaErrors(hipEventCreate(&wp_event));

    size_t records = 0;

    gettimeofday(&t1, 0);

    for (int s = 0; s < steps; s++) {
        ////////////////////////////////////////////////////////////////////////////////////////////////////////////////
        //                                       FILL IN: kernels invocation (step 0)                                 //
        ////////////////////////////////////////////////////////////////////////////////////////////////////////////////

        calculate_velocity<<< simulationGrid, thr_blc, shm_mem_calc, cp_stream >>>
            (particles_gpu[s & 1ul], particles_gpu[(s + 1) & 1ul], N, dt);
        checkCudaErrors(hipEventRecord(cp_event, cp_stream));

        checkCudaErrors(hipMemcpyAsync(
            particles_cpu.pos_x, particles_gpu[s & 1ul].pos_x, size, hipMemcpyDeviceToHost, wp_stream
        ));
        checkCudaErrors(hipMemcpyAsync(
            particles_cpu.pos_y, particles_gpu[s & 1ul].pos_y, size, hipMemcpyDeviceToHost, wp_stream
        ));
        checkCudaErrors(hipMemcpyAsync(
            particles_cpu.pos_z, particles_gpu[s & 1ul].pos_z, size, hipMemcpyDeviceToHost, wp_stream
        ));
        checkCudaErrors(hipMemcpyAsync(
            particles_cpu.vel_x, particles_gpu[s & 1ul].vel_x, size, hipMemcpyDeviceToHost, wp_stream
        ));
        checkCudaErrors(hipMemcpyAsync(
            particles_cpu.vel_y, particles_gpu[s & 1ul].vel_y, size, hipMemcpyDeviceToHost, wp_stream
        ));
        checkCudaErrors(hipMemcpyAsync(
            particles_cpu.vel_z, particles_gpu[s & 1ul].vel_z, size, hipMemcpyDeviceToHost, wp_stream
        ));
        checkCudaErrors(hipMemcpyAsync(
            particles_cpu.weight, particles_gpu[s & 1ul].weight, size, hipMemcpyDeviceToHost, wp_stream
        ));

        ////////////////////////////////////////////////////////////////////////////////////////////////////////////////
        //                                          FILL IN: synchronization  (step 4)                                //
        ////////////////////////////////////////////////////////////////////////////////////////////////////////////////

        checkCudaErrors(hipMemsetAsync(comGPU, 0, sizeof(float4), cm_stream));
        compute_gpu_center_of_mass(
                particles_gpu[s & 1ul], &comGPU[0], &lock[0], N, reductionGrid, red_thr_blc, shm_mem_mass, &cm_stream
        );
        checkCudaErrors(hipMemcpyAsync(comCPU, comGPU, sizeof(float4), hipMemcpyDeviceToHost, cm_stream));

        if (writeFreq > 0 && (s % writeFreq == 0)) {
            ////////////////////////////////////////////////////////////////////////////////////////////////////////////
            //                          FILL IN: synchronization and file access logic (step 4)                       //
            ////////////////////////////////////////////////////////////////////////////////////////////////////////////
            // Writing final values to the file
            checkCudaErrors(hipStreamSynchronize(cm_stream));
            h5Helper.writeCom(
                comCPU[0].x / comCPU[0].w,
                comCPU[0].y / comCPU[0].w,
                comCPU[0].z / comCPU[0].w,
                comCPU[0].w, records
            );

            checkCudaErrors(hipStreamSynchronize(wp_stream));
            h5Helper.writeParticleData(records++);
        }

//        checkCudaErrors(hipStreamSynchronize(cp_stream));
        checkCudaErrors(hipStreamWaitEvent(wp_stream, cp_event, 0));
        checkCudaErrors(hipStreamWaitEvent(cm_stream, cp_event, 0));
    }

    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    //              FILL IN: invocation of center-of-mass kernel (step 3.1, step 3.2, step 4)                         //
    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

    hipDeviceSynchronize();

    checkCudaErrors(hipMemset(comGPU, 0, sizeof(float4)));
    compute_gpu_center_of_mass(
            particles_gpu[steps & 1], &comGPU[0], &lock[0], N, reductionGrid, red_thr_blc, shm_mem_mass, &cm_stream
    );
    checkCudaErrors(hipMemcpy(comCPU, comGPU, sizeof(float4), hipMemcpyDeviceToHost));

    gettimeofday(&t2, 0);

    // Approximate simulation wall time
    double t = (1000000.0 * (t2.tv_sec - t1.tv_sec) + t2.tv_usec - t1.tv_usec) / 1000000.0;
    printf("Time: %f s\n", t);

    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    //                             FILL IN: memory transfers for particle data (step 0)                               //
    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

    checkCudaErrors(hipMemcpy(particles_cpu.pos_x, particles_gpu[steps & 1].pos_x, size, hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(particles_cpu.pos_y, particles_gpu[steps & 1].pos_y, size, hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(particles_cpu.pos_z, particles_gpu[steps & 1].pos_z, size, hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(particles_cpu.vel_x, particles_gpu[steps & 1].vel_x, size, hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(particles_cpu.vel_y, particles_gpu[steps & 1].vel_y, size, hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(particles_cpu.vel_z, particles_gpu[steps & 1].vel_z, size, hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(particles_cpu.weight, particles_gpu[steps & 1].weight, size, hipMemcpyDeviceToHost));

    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    //                        FILL IN: memory transfers for center-of-mass (step 3.1, step 3.2)                       //
    ////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
    float4 comOnCPU = centerOfMassCPU(md);

    std::cout << "Center of mass on CPU:" << std::endl
              << comOnCPU.x << ", "
              << comOnCPU.y << ", "
              << comOnCPU.z << ", "
              << comOnCPU.w
              << std::endl;

    std::cout << "Center of mass on GPU:" << std::endl
              << comCPU[0].x / comCPU[0].w  << ", "
              << comCPU[0].y / comCPU[0].w << ", "
              << comCPU[0].z / comCPU[0].w << ", "
              << comCPU[0].w
              << std::endl;

    // Writing final values to the file
    h5Helper.writeComFinal(
        comCPU[0].x / comCPU[0].w,
        comCPU[0].y / comCPU[0].w,
        comCPU[0].z / comCPU[0].w,
        comCPU[0].w / comCPU[0].w
    );
    h5Helper.writeParticleDataFinal();

    checkCudaErrors(hipStreamDestroy(cp_stream));
    checkCudaErrors(hipStreamDestroy(wp_stream));
    checkCudaErrors(hipStreamDestroy(cm_stream));

    checkCudaErrors(hipHostFree(particles_cpu.pos_x));
    checkCudaErrors(hipHostFree(particles_cpu.pos_y));
    checkCudaErrors(hipHostFree(particles_cpu.pos_z));
    checkCudaErrors(hipHostFree(particles_cpu.vel_x));
    checkCudaErrors(hipHostFree(particles_cpu.vel_y));
    checkCudaErrors(hipHostFree(particles_cpu.vel_z));
    checkCudaErrors(hipHostFree(particles_cpu.weight));
    checkCudaErrors(hipHostFree(comCPU));

    for (auto p_gpu : particles_gpu) {
        checkCudaErrors(hipFree(p_gpu.pos_x));
        checkCudaErrors(hipFree(p_gpu.pos_y));
        checkCudaErrors(hipFree(p_gpu.pos_z));
        checkCudaErrors(hipFree(p_gpu.vel_x));
        checkCudaErrors(hipFree(p_gpu.vel_y));
        checkCudaErrors(hipFree(p_gpu.vel_z));
        checkCudaErrors(hipFree(p_gpu.weight));
    }
    checkCudaErrors(hipFree(comGPU));
    checkCudaErrors(hipFree(lock));

    return 0;
}// end of main
//----------------------------------------------------------------------------------------------------------------------
